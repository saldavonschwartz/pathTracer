#include "hip/hip_runtime.h"
//
//  Sphere.cpp
//  PathTracer
//
//  Created by Federico Saldarini on 4/16/20.
//  Copyright © 2020 Federico Saldarini. All rights reserved.
//

#include "Sphere.hpp"
#include "Material.hpp"

__device__ Sphere::Sphere(gvec3 position, float radius, Material* material)
	: position(position), radius(radius), material(material) {}

__device__ Sphere::~Sphere() {
	delete material;
}

__device__ bool Sphere::boundingBox(double t0, double t1, AABA& bBox) const {
	gvec3 extent{ radius };
	bBox = AABA(position - extent, position + extent);
	return true;
}

__device__ bool Sphere::hit(const Ray& ray, float tmin, float tmax, HitInfo& info) const {
	// From ray eq. r(t) = 0 + t*d and sphere eq. (x-c)^2 - r^2 = 0
	// Solve quadratic (r(t)-c)^2 -r^2 = 0 -> a*t^2 + b*t + c = 0
	// 0 roots = no hit, 1 root = tanget hit, 2 roots = went in and thru:

	gvec3 oc = ray.origin - position;
	float a = dot(ray.dir, ray.dir);
	float b = dot(oc, ray.dir);
	float c = dot(oc, oc) - radius * radius;
	float discriminant = b * b - a * c;

	if (discriminant > 0) {
		float t = (-b - sqrt(discriminant)) / a;

		if (tmin < t && t < tmax) {
			auto hitPoint = ray(t);
			info.hitPoint = hitPoint;
			info.normal = (hitPoint - position) / radius;
			info.isFrontFace = dot(info.normal, ray.dir) < 0.f;
			info.normal = info.isFrontFace ? info.normal : -info.normal;
			info.t = t;
			info.material = material;
			return true;
		}

		t = (-b + sqrt(discriminant)) / a;

		if (tmin < t && t < tmax) {
			auto hitPoint = ray(t);
			info.hitPoint = hitPoint;
			info.normal = (hitPoint - position) / radius;
			info.isFrontFace = dot(info.normal, ray.dir) < 0.f;
			info.normal = info.isFrontFace ? info.normal : -info.normal;
			info.t = t;
			info.material = material;
			return true;
		}
	}

	return false;
}