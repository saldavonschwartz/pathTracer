//
//  Material.cpp
//  PathTracer
//
//  Created by Federico Saldarini on 4/16/20.
//  Copyright © 2020 Federico Saldarini. All rights reserved.
//

#include "Material.hpp"
#include "Utils.hpp"

__device__ bool Diffuse::scatter(const Ray& ray, const HitInfo& info, gvec3& attenuation, Ray& scattered, hiprandState* rState) const {
  gvec3 scatterDir = info.normal + sphericalRand(1.f, rState);
  scattered = Ray(info.hitPoint, scatterDir);
  attenuation = albedo;
  return true;
}

__device__ bool Metal::scatter(const Ray& ray, const HitInfo& info, gvec3& attenuation, Ray& scattered, hiprandState* rState) const {
  gvec3 scatterDir = reflect(ray.dir, info.normal);
  scattered = Ray(info.hitPoint, scatterDir + fuzziness * ballRand(1.f, rState));
  attenuation = albedo;
  return dot(scattered.dir, info.normal) > 0.f;
}

__device__ bool Dielectric::scatter(const Ray& ray, const HitInfo& info, gvec3& attenuation, Ray& scattered, hiprandState* rState) const {
  attenuation = gvec3{1};
  auto k1 = info.isFrontFace ? 1.f : refractionIdx;
  auto k2 = info.isFrontFace ? refractionIdx : 1.f;
  auto r = reflectance(ray.dir, info.normal, k1, k2);
  
  if (r == 1.f || hiprand_uniform(rState) < r) {
    gvec3 scatterDir = reflect(ray.dir, info.normal);
    scattered = Ray(info.hitPoint, scatterDir);
    return true;
  }
  
  gvec3 scatterDir = refract(ray.dir, info.normal, k1, k2);
  scattered = Ray(info.hitPoint, scatterDir);
  return true;
}
