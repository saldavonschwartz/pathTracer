#include "hip/hip_runtime.h"
//
//  Camera.cpp
//  PathTracer
//
//  Created by Federico Saldarini on 4/16/20.
//  Copyright © 2020 Federico Saldarini. All rights reserved.
//

#include "Camera.hpp"
#include "Utils.hpp"


__device__ Ray Camera::castRay(float u, float v, hiprandState* rs) const {
	auto r = (a / 2.f) * diskRand(1.f, rs);
	auto offset = x * r.x + y * r.y;
	auto dir = lowerLeftImageOrigin + u * wOffset + v * hOffset - position - offset;
	return { position + offset, dir };
}

__device__ void Camera::init(
	const gvec3& position, const gvec3& lookAt, float fovy,
	float aspect, float flLength, float aperture
)
{
	this->position = position;
	this->aspect = aspect;
	this->f = flLength;
	this->a = aperture;
	this->fovy = fovy;

	float hh = tanf((fovy * (pi / 180.f) / 2.f));
	float hw = aspect * hh;

	// [x y z p] = new camera orientation:
	gvec3 z = normalize(position - lookAt);
	x = normalize(cross({ 0.f, 1.f, 0.f }, z));
	y = cross(z, x);

	auto& f = flLength;
	lowerLeftImageOrigin = position - x * hw*f - y * hh*f - z * f;
	hOffset = 2 * hh*f*y;
	wOffset = 2 * hw*f*x;
}
