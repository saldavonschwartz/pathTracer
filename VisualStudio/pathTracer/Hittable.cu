#include "hip/hip_runtime.h"
//
//  Hittable.cpp
//  PathTracer
//
//  Created by Federico Saldarini on 4/16/20.
//  Copyright © 2020 Federico Saldarini. All rights reserved.
//

#include "Hittable.hpp"

__device__ AABA AABA::surroundingBox(const AABA& b0, const AABA& b1) {
	gvec3 small = {
		fminf(b0.xmin.x, b1.xmin.x),
		fminf(b0.xmin.y, b1.xmin.y) ,
		fminf(b0.xmin.z, b1.xmin.z)
	};

	gvec3 big = {
		fmaxf(b0.xmax.x, b1.xmax.x),
		fmaxf(b0.xmax.y, b1.xmax.y),
		fmaxf(b0.xmax.z, b1.xmax.z)
	};

	return { small, big };
}

__device__ AABA::AABA() {};

__device__ AABA::AABA(gvec3 xmin, gvec3 xmax) : xmin(xmin), xmax(xmax) {}

__device__ bool AABA::hit(const Ray& ray, float tmin, float tmax) const {
	for (int a = 0; a < 3; a++) {
		auto invD = 1.0f / ray.dir[a];
		auto t0 = (xmin[a] - ray.origin[a]) * invD;
		auto t1 = (xmax[a] - ray.origin[a]) * invD;

		if (invD < 0.0f) {
			auto tn = t0;
			t0 = t1;
			t1 = tn;
		}

		tmin = t0 > tmin ? t0 : tmin;
		tmax = t1 < tmax ? t1 : tmax;

		if (tmax <= tmin) {
			return false;
		}
	}

	return true;
}