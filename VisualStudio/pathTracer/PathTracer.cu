#include "hip/hip_runtime.h"
//
//  PathTracer.cpp
//  PathTracer
//
//  Created by Federico Saldarini on 4/16/20.
//  Copyright © 2020 Federico Saldarini. All rights reserved.
//

#include "PathTracer.hpp"

#include <iostream>
#include <iomanip>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "Utils.hpp"
#include "Ray.hpp"
#include "Material.hpp"
#include "Camera.hpp"
#include "HittableVector.hpp"
#include "BVH.hpp"
#include "Sphere.hpp"

using std::ofstream;
using std::string;
using std::cerr;

__device__ gvec3 sampleRay(Ray ray, float tmin, float tmax, int maxBounces, Hittable* hittable,  hiprandState* rState) {
	gvec3 color{1.f};
	gvec3 attenuation;

	while (maxBounces > -1) {
		Hittable::HitInfo info;

		if (hittable->hit(ray, tmin, tmax, info)) {		
			if (info.material->scatter(ray, info, attenuation, ray, rState)) {
				color *= attenuation;
				maxBounces -= 1;
			}
			else {
				color *= {};
				break;
			}
		}
		else {
			float t = 0.5f * (ray.dir + gvec3{ 1.f, 1.f, 1.f }).y;
			color *= (1.f - t) * gvec3 { 1.f, 1.f, 1.f } +t * gvec3{ .5f, .7f, 1.f };
			break;
		}
	}

	return color;
}

__global__ void generateSimpleScene(HittableVector** scene, Camera* cam, float aspect) {
	*scene = new HittableVector();
	(*scene)->init(5);
	(*scene)->add(new Sphere(gvec3(0.f, 0.f, -1.f), 0.5f, new Diffuse(gvec3(0.1f, 0.2f, 0.5f))));
	(*scene)->add(new Sphere(gvec3(0.f, -100.5f, -1.f), 100.f, new Diffuse(gvec3(0.8f, 0.8f, 0.0f))));
	(*scene)->add(new Sphere(gvec3(1.f, 0.f, -1.f), 0.5f, new Metal(gvec3(0.8f, 0.6f, 0.2f), 0.3f)));
	(*scene)->add(new Sphere(gvec3(-1.f, 0.f, -1.f), 0.5f, new Dielectric(1.5f)));
	(*scene)->add(new Sphere(gvec3(-1.f, 0.f, -1.f), -0.45f, new Dielectric(1.5f)));

	gvec3 lookFrom = { 3.f, 3.f, 2.f };
	gvec3 lookAt = { 0.f, 0.f, -1.f };
	float f = length(lookFrom - lookAt);
	float a = 0.1f;
	float fovy = 20.f;
	cam->init(lookFrom, lookAt, fovy, aspect, f, a);
}

__global__ void generateSimpleScene2(HittableVector* scene, Camera* cam, float aspect) {
	scene->init(5);
	scene->add(new Sphere(gvec3(0.f, 0.f, -1.f), 0.5f, new Diffuse(gvec3(0.1f, 0.2f, 0.5f))));
	scene->add(new Sphere(gvec3(0.f, -100.5f, -1.f), 100.f, new Diffuse(gvec3(0.8f, 0.8f, 0.0f))));
	scene->add(new Sphere(gvec3(1.f, 0.f, -1.f), 0.5f, new Metal(gvec3(0.8f, 0.6f, 0.2f), 0.3f)));
	scene->add(new Sphere(gvec3(-1.f, 0.f, -1.f), 0.5f, new Dielectric(1.5f)));
	scene->add(new Sphere(gvec3(-1.f, 0.f, -1.f), -0.45f, new Dielectric(1.5f)));

	gvec3 lookFrom = { 3.f, 3.f, 2.f };
	gvec3 lookAt = { 0.f, 0.f, -1.f };
	float f = length(lookFrom - lookAt);
	float a = 0.1f;
	float fovy = 20.f;
	cam->init(lookFrom, lookAt, fovy, aspect, f, a);
}

__global__ void generateComplexScene(
	BVHNode** bvh, HittableVector** scene, Camera* cam, float aspect, hiprandState* rState) {
	if (threadIdx.x != 0 || blockIdx.x != 0) {
		return;
	}

	hiprandState rs = *rState;
	int x = 10, y = 10;
	int size = ((x * 2) * (y * 2)) + 4;
	
	*scene = new HittableVector();
	(*scene)->init(size);
	
	(*scene)->add(new Sphere(gvec3{ 0.f,-1000.f,0.f }, 1000.f, new Diffuse(gvec3{ 0.5f })));
	(*scene)->add(new Sphere(gvec3{ 0.f, 1.f, 0.f }, 1.f, new Dielectric(1.5f)));
	(*scene)->add(new Sphere(gvec3{ -4.f, 1.f, 0.f }, 1.f, new Diffuse(gvec3{ 0.4f, 0.2f, 0.1f })));
	(*scene)->add(new Sphere(gvec3{ 4.f, 1.f, 0.f }, 1.f, new Metal(gvec3{ 0.7f, 0.6f, 0.5f }, 0.f)));

	for (int a = -x; a < x; a++) {
		for (int b = -y; b < y; b++) {
			auto materialProbability = hiprand_uniform(&rs);
			gvec3 center{ a + 0.9f * hiprand_uniform(&rs), 0.2f, b + 0.9f * hiprand_uniform(&rs) };

			if (length(center - gvec3{ 4.f, 0.2f, 0.f }) > 0.9f) {
				if (materialProbability < 0.8f) {
					// Diffuse
					gvec3 albedo = urand3(&rs) * urand3(&rs);
					(*scene)->add(new Sphere(center, 0.2f, new Diffuse(albedo)));
				}
				else if (materialProbability < 0.95f) {
					// Metal
					gvec3 albedo = (urand3(&rs) + 1.f) * 0.5f;
					auto fuzziness = hiprand_uniform(&rs) * 0.5f;
					(*scene)->add(new Sphere(center, 0.2f, new Metal(albedo, fuzziness)));
				}
				else {
					// glass
					(*scene)->add(new Sphere(center, 0.2f, new Dielectric(1.5f)));
				}
			}
		}
	}
	
	//*bvh = new BVHNode(*scene, 0.f, 1.f, &rs);
	*rState = rs;

	gvec3 lookFrom = { 13.f, 2.f, 3.f };
	gvec3 lookAt = { 0.f };
	float f = 10.f;
	float a = 0.1f;
	float fovy = 20.f;

	cam->init(lookFrom, lookAt, fovy, aspect, f, a);
}

__global__ void randInit(int w, int h, hiprandState* randState) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= w || y >= h) {
		return;
	}

	int p = y * w + x;
	hiprand_init(1984+p, 0, 0, &randState[p]);
}

__global__ void randInit1(hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void renderScene(
	gvec3* frameBuff, int w, int h, Camera* cam, Hittable** scene,
	int raysPerPixel, int maxBouncesPerRay, hiprandState* rStates
)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= w || y >= h) {
		return;
	}

	int p = y * w + x;
	hiprandState rs = rStates[p];
	gvec3 pixel;

	for (int r = 0; r < raysPerPixel; r++) {
		float u = (x + hiprand_uniform(&rs)) / float(w);
		float v = (y + hiprand_uniform(&rs)) / float(h);
		Ray ray = cam->castRay(u, v, &rs);
		pixel += sampleRay(ray, 0.001f, inf, maxBouncesPerRay, *scene, &rs);
	}

	rStates[p] = rs;

	pixel /= float(raysPerPixel);
	pixel.r = sqrt(pixel.r);
	pixel.g = sqrt(pixel.g);
	pixel.b = sqrt(pixel.b);
	frameBuff[p] = 255.f * pixel;
}

__global__ void freeScene(Hittable** scene) {
	delete (*scene);
}

int renderScene(int sceneId, string path, int width, int height, int raysPerPixel, int maxBouncesPerRay) {
	ofstream outputImage(path + "imgOutCuda.ppm");

	if (!outputImage.is_open()) {
		cerr << "ERROR: could not open output file!\n";
		return -1;
	}

	float aspect = float(width) / height;
	size_t pixelCount = width * height;
	dim3 threads(8, 8);
	dim3 blocks(width / threads.x + 1, height / threads.y + 1);

	hipDeviceReset();

	// One random to generate the scene:
	hiprandState *sceneGenRand;
	CHK_CUDA(hipMalloc(&sceneGenRand, sizeof(hiprandState)));
	randInit1 << <1, 1 >> > (sceneGenRand);
	CHK_CUDA(hipGetLastError());
	CHK_CUDA(hipDeviceSynchronize());

	// w * h randoms to render (one per pixel):
	hiprandState *perPixelRand;
	CHK_CUDA(hipMalloc(&perPixelRand, sizeof(hiprandState) * pixelCount));
	randInit << <blocks, threads >> > (width, height, perPixelRand);
	CHK_CUDA(hipGetLastError());
	CHK_CUDA(hipDeviceSynchronize());

	// Generate scene + camera:
	HittableVector** scene;
	CHK_CUDA(hipMalloc(&scene, sizeof(HittableVector*)));

	BVHNode** bvh;
	CHK_CUDA(hipMalloc(&bvh, sizeof(BVHNode*)));

	Camera* cam;
	CHK_CUDA(hipMalloc(&cam, sizeof(Camera)));

	 //generateSimpleScene2 << <1, 1 >> > (scene, cam, aspect);
	generateComplexScene << <1, 1 >> > (bvh, scene, cam, aspect, sceneGenRand);
	CHK_CUDA(hipGetLastError());
	CHK_CUDA(hipDeviceSynchronize());
	//BVHNode scene(sceneObjects, 0.f, 1.f);
	
	// alloc frame buffer:
	gvec3* fBuffer;
	CHK_CUDA(hipMallocManaged(&fBuffer, sizeof(float) * 3 * pixelCount));
	
	// Render:
	renderScene << <blocks, threads >> > (fBuffer, width, height, cam, (Hittable**)scene, raysPerPixel, maxBouncesPerRay, perPixelRand);	
	CHK_CUDA(hipGetLastError());
		
	{
		auto p = Profiler("[Render Time]");
		CHK_CUDA(hipDeviceSynchronize());
	}

	{
		auto p = Profiler("[Save Time]");

		// Save file as ascii PPM:
		// http://netpbm.sourceforge.net/doc/ppm.html#plainppm
		outputImage << "P3\n" << width << " " << height << "\n255\n";

		// Image origin is bottom-left.
		// Pixels are output one row at a time, top to bottom, left to right:

		for (int y = height - 1; y >= 0; y--) {
			for (int x = 0; x < width; x++) {
				int p = y * width + x;
				gvec3 pixel = fBuffer[p];
				outputImage
					<< int(pixel.r) << " "
					<< int(pixel.g) << " "
					<< int(pixel.b) << "\n";
			}
		}

		outputImage.close();
	}

	freeScene << <1, 1 >> > ((Hittable**)scene);
	CHK_CUDA(hipGetLastError());
	CHK_CUDA(hipDeviceSynchronize());

	CHK_CUDA(hipFree(fBuffer));
	CHK_CUDA(hipFree(scene));
	CHK_CUDA(hipFree(cam));
	return 0;
}



